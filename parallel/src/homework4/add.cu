
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1 << 20; // 1M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;
  // Run kernel on 1M elements on the CPU
  add<<<num_blocks, block_size>>>(N, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  printf("Max error: %.02f\n", maxError);

  hipFree(x);
  hipFree(y);

  return 0;
}